/**
 * @file db.cu
 * @author Yihao Sun (ysun799@bloomberg.com)
 * @brief GPU clause database
 *
 * This file is a GPU version of struct Yals in yals.c
 *
 * @version 0.1
 * @date 2025-06-05
 *
 * @copyright Copyright (c) 2025
 */

#include "gpu/gpu.h"
#include "gpu/yalsg.cuh"
#include "rmm/device_vector.hpp"
#include "yils.h"

#include "rmm/mr/device/per_device_resource.hpp"
#include <cstdint>
#include <rmm/mr/device/cuda_memory_resource.hpp>

#include <stdint.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>

GPUMemoryManager::GPUMemoryManager() {
  mr = new rmm::mr::cuda_memory_resource();
  auto initial_size = rmm::percent_of_free_device_memory(25);
  pool = new rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource>(
      mr, initial_size);
  rmm::mr::set_current_device_resource(pool);
}

GPUMemoryManager *gpu_memory_manager_new() { return new GPUMemoryManager(); }

Yalsg *yalsg_new(Yals *yals) { return new Yalsg(yals); }

void yalsg_load_to_device(Yalsg *yalsg) {
  if (!yalsg->yals) {
    throw std::runtime_error("Yalsg is not initialized");
  }

  yalsg->unsat.resize(yalsg->yals->nvars);
  if (yalsg->yals->unsat.usequeue) {
    // throw error for unimplemented queue
    throw std::runtime_error("Queue is not implemented on GPU");
  } else {
    int cnt = SIZE(yalsg->yals->unsat.stack);
    hipMemcpy(yalsg->unsat.data().get(), yalsg->yals->unsat.stack.start,
               cnt * sizeof(int), hipMemcpyHostToDevice);
  }

  // load occ stack into device
  yalsg->occs.resize(yalsg->yals->noccs);
  hipMemcpy(yalsg->occs.data().get(), yalsg->yals->occs,
             yalsg->yals->noccs * sizeof(int), hipMemcpyHostToDevice);

  // load refs into device, unlike host refs, device refs will also store the
  // size of each occ stack bucket
  yalsg->refs.resize(2 * yalsg->yals->nvars);
  hipMemcpy(yalsg->refs.data().get(), yalsg->yals->refs,
             2 * yalsg->yals->nvars * sizeof(int), hipMemcpyHostToDevice);
  yalsg->occs_len.resize(2 * yalsg->yals->nvars);
  hipMemcpy(yalsg->occs_len.data().get(), yalsg->yals->occs_bucket_sizes,
             2 * yalsg->yals->nvars * sizeof(uint32_t), hipMemcpyHostToDevice);
  
  // locad flips into device
  yalsg->flips.resize(yalsg->yals->nvars);
  hipMemcpy(yalsg->flips.data().get(), yalsg->yals->flips,
             yalsg->yals->nvars * sizeof(uint32_t), hipMemcpyHostToDevice);

  // load vals into device
  yalsg->vals.resize(yalsg->yals->nvarwords);
  hipMemcpy(yalsg->vals.data().get(), yalsg->yals->vals,
             yalsg->yals->nvarwords * sizeof(Word), hipMemcpyHostToDevice);
  // words
  // load best into device
  yalsg->best.resize(yalsg->yals->nvarwords);
  hipMemcpy(yalsg->best.data().get(), yalsg->yals->best,
             yalsg->yals->nvarwords * sizeof(Word), hipMemcpyHostToDevice);
  // load tmp into device
  yalsg->tmp.resize(yalsg->yals->nvarwords);
  hipMemcpy(yalsg->tmp.data().get(), yalsg->yals->tmp,
             yalsg->yals->nvarwords * sizeof(Word), hipMemcpyHostToDevice);
  // load clear into device
  yalsg->clear.resize(yalsg->yals->nvarwords);
  hipMemcpy(yalsg->clear.data().get(), yalsg->yals->clear,
             yalsg->yals->nvarwords * sizeof(Word), hipMemcpyHostToDevice);
  // load sets into device
  yalsg->sets.resize(yalsg->yals->nvarwords);
  hipMemcpy(yalsg->sets.data().get(), yalsg->yals->set,
             yalsg->yals->nvarwords * sizeof(Word), hipMemcpyHostToDevice);
  
  auto size_of_cdb = SIZE(yalsg->yals->cdb);
  yalsg->cdb.resize(size_of_cdb);
  hipMemcpy(yalsg->cdb.data().get(), yalsg->yals->cdb.start,
             size_of_cdb * sizeof(int), hipMemcpyHostToDevice);
  // load cache into device
  // yalsg->cache.resize(yalsg->yals->cachesizetarget);
  // hipMemcpy(yalsg->cache.data().get(), yalsg->yals->cache,
  //            yalsg->yals->cachesizetarget * sizeof(Word), hipMemcpyHostToDevice);
  
  // load mins into device
  auto size_of_mins = SIZE(yalsg->yals->mins);
  yalsg->mins.resize(size_of_mins);
  hipMemcpy(yalsg->mins.data().get(), yalsg->yals->mins.start,
             size_of_mins * sizeof(int), hipMemcpyHostToDevice);
  
  // load satcnt into device
  yalsg->satcnt.resize(yalsg->yals->nvars);
  hipMemcpy(yalsg->satcnt.data().get(), yalsg->yals->satcnt4,
             yalsg->yals->nvars * sizeof(uint32_t), hipMemcpyHostToDevice);

  // load weights into device
  yalsg->weights.resize(yalsg->yals->nvars);
  hipMemcpy(yalsg->weights.data().get(), yalsg->yals->weights,
             yalsg->yals->nvars * sizeof(uint32_t), hipMemcpyHostToDevice);
  // load weightedbreak into device
  yalsg->weightedbreak.resize(yalsg->yals->nvars);
  hipMemcpy(yalsg->weightedbreak.data().get(), yalsg->yals->weightedbreak,
             yalsg->yals->nvars * sizeof(uint32_t), hipMemcpyHostToDevice);
  
  // load crit into device
  yalsg->crit.resize(yalsg->yals->nclauses);
  hipMemcpy(yalsg->crit.data().get(), yalsg->yals->crit,
             yalsg->yals->nclauses * sizeof(int), hipMemcpyHostToDevice);
  
  // load breaks into device
  auto size_of_breaks = SIZE(yalsg->yals->breaks);
  yalsg->breaks.resize(size_of_breaks);
  hipMemcpy(yalsg->breaks.data().get(), yalsg->yals->breaks.start,
             size_of_breaks * sizeof(uint32_t), hipMemcpyHostToDevice);
  
  // load scores into device
  auto size_of_scores = SIZE(yalsg->yals->scores);
  yalsg->scores.resize(size_of_scores);
  hipMemcpy(yalsg->scores.data().get(), yalsg->yals->scores.start,
             size_of_scores * sizeof(double), hipMemcpyHostToDevice);
  
  // load cands into device
  auto size_of_cands = SIZE(yalsg->yals->cands);
  yalsg->cands.resize(size_of_cands);
  hipMemcpy(yalsg->cands.data().get(), yalsg->yals->cands.start,
             size_of_cands * sizeof(int), hipMemcpyHostToDevice);
  
  // TODO: check all cache related vectors ds
}
